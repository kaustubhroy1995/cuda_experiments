
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel(){
  printf("Hello!");
}

using namespace std;
int main(){
  kernel<<< 1, 32 >>>();
  hipDeviceSynchronize();
}
